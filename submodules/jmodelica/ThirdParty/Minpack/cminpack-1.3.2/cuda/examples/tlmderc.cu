#include "hip/hip_runtime.h"
/* -*- mode: c++ -*- */
/* ------------------------------ */
/*  driver for lmder example.     */
/* ------------------------------ */

#include <stdio.h>
#include <math.h>
#include <string.h>
#include <cminpack.h>

#include <lmder.cu>
#include <covar1.cu>
#define real __cminpack_real__

#define cutilSafeCall(err)           __cudaSafeCall      (err, __FILE__, __LINE__)
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    if( hipSuccess != err) {
        fprintf(stderr, "cudaSafeCall() Runtime API error in file <%s>, line %i : %s.\n",
                file, line, hipGetErrorString( err) );
        exit(-1);
    }
}

//--------------------------------------------------------------------------
//--------------------------------------------------------------------------
const unsigned int NUM_OBSERVATIONS = 15; // m
const unsigned int NUM_PARAMS = 3; // 3 = n

//--------------------------------------------------------------------------
//--------------------------------------------------------------------------
// 
//  fixed arrangement of threads to be run 
// 
const unsigned int NUM_THREADS = 2048;
const unsigned int NUM_THREADS_PER_BLOCK = 128;
const unsigned int NUM_BLOCKS = NUM_THREADS / NUM_THREADS_PER_BLOCK;

//--------------------------------------------------------------------------
//
// the struct for returning results from the GPU
//

typedef struct    
{
    real fnorm;
    int nfev;
    int njev;
    int info;
    int rankJ;
    real solution[NUM_PARAMS];
    real covar[NUM_PARAMS][NUM_PARAMS];
} ResultType;

//--------------------------------------------------------------------------
// the cost function
//--------------------------------------------------------------------------
__cminpack_attr__ /* __device__ */
int fcnder_mn(
    void *p, int m, int n, const real *x, 
    real *fvec, real *fjac, 
    int ldfjac, int iflag)
{

    /*      subroutine fcn for lmder example. */

    int i;
    real tmp1, tmp2, tmp3, tmp4;
    real y[NUM_OBSERVATIONS]={1.4e-1, 1.8e-1, 2.2e-1, 2.5e-1, 
                              2.9e-1, 3.2e-1, 3.5e-1, 3.9e-1, 3.7e-1, 
                              5.8e-1, 7.3e-1, 9.6e-1, 1.34, 2.1, 4.39};

    if (iflag == 0) {
        /*      insert print statements here when nprint is positive. */
        return 0;
    }

    if (iflag != 2) {

	for (i = 1; i <= NUM_OBSERVATIONS; i++) {
            tmp1 = i;
            tmp2 = (NUM_OBSERVATIONS+1) - i;
            tmp3 = tmp1;
            if (i > 8) tmp3 = tmp2;
            fvec[i-1] = y[i-1] - (x[1-1] + tmp1/(x[2-1]*tmp2 + x[3-1]*tmp3));
        } // for

    } else { 

        for (i=1; i<=NUM_OBSERVATIONS; i++) {
            tmp1 = i;
            tmp2 = (NUM_OBSERVATIONS+1) - i;
            tmp3 = tmp1;
            if (i > 8) tmp3 = tmp2;
            tmp4 = (x[2-1]*tmp2 + x[3-1]*tmp3); tmp4 = tmp4*tmp4;
            fjac[i-1 + ldfjac*(1-1)] = -1.;
            fjac[i-1 + ldfjac*(2-1)] = tmp1*tmp2/tmp4;
            fjac[i-1 + ldfjac*(3-1)] = tmp1*tmp3/tmp4;
        } // for
    } // if

    return 0;
}

//--------------------------------------------------------------------------
// the kernel in the GPU
//--------------------------------------------------------------------------
__global__ void mainKernel(ResultType  pResults[])
{
    int ldfjac, maxfev, mode, nprint, info, nfev, njev;
    int ipvt[NUM_PARAMS];
    real ftol, xtol, gtol, factor, fnorm;
    real x[NUM_PARAMS], fvec[NUM_OBSERVATIONS], 
            diag[NUM_PARAMS], fjac[NUM_OBSERVATIONS*NUM_PARAMS], qtf[NUM_PARAMS], 
            wa1[NUM_PARAMS], wa2[NUM_PARAMS], wa3[NUM_PARAMS], wa4[NUM_OBSERVATIONS];
    int k;

    // m = NUM_OBSERVATIONS;
    // n = NUM_PARAMS;

    /*      the following starting values provide a rough fit. */

    x[1-1] = 1.; 
    x[2-1] = 1.; 
    x[3-1] = 1.;

    ldfjac = NUM_OBSERVATIONS;

    /*      set ftol and xtol to the square root of the machine */
    /*      and gtol to zero. unless high solutions are */
    /*      required, these are the recommended settings. */

    ftol = sqrt(__cminpack_func__(dpmpar)(1));
    xtol = sqrt(__cminpack_func__(dpmpar)(1));
    gtol = 0.;

    maxfev = 400;
    mode = 1;
    factor = 1.e2;
    nprint = 0;

    // -------------------------------
    // call lmder, enorm, and covar1
    // -------------------------------
    info = __cminpack_func__(lmder)(__cminpack_param_fcnder_mn__ 0, NUM_OBSERVATIONS, NUM_PARAMS, 
                 x, fvec, fjac, ldfjac, ftol, xtol, gtol, 
                 maxfev, diag, mode, factor, nprint, &nfev, &njev, 
                 ipvt, qtf, wa1, wa2, wa3, wa4);

    fnorm = __cminpack_func__(enorm)(NUM_OBSERVATIONS, fvec);

    // NOTE: REMOVED THE TEST OF ORIGINAL MINPACK covar routine

    /* test covar1, which also estimates the rank of the Jacobian */
    ftol = __cminpack_func__(dpmpar)(1);
    k = __cminpack_func__(covar1)(NUM_OBSERVATIONS, NUM_PARAMS, 
               fnorm*fnorm, fjac, ldfjac, ipvt, ftol, wa1);

    // ----------------------------------
    // save the results in global memory
    // ----------------------------------
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;

    pResults[threadId].fnorm = fnorm;
    pResults[threadId].nfev = nfev;
    pResults[threadId].njev = njev;
    pResults[threadId].info = info;

    for (int j=1; j<=NUM_PARAMS; j++) {
        pResults[threadId].solution[j-1] = x[j-1];
    }

    for (int i=1; i<=NUM_PARAMS; i++) {
        for (int j=1; j<=NUM_PARAMS; j++) {
            pResults[threadId].covar[i-1][j-1] = fjac[(i-1)*ldfjac+j-1];
	} // for
    } // for

    pResults[threadId].rankJ =  (k != 0 ? k : NUM_PARAMS); 

} // ()

//--------------------------------------------------------------------------
//--------------------------------------------------------------------------
int main (int argc, char** argv)
{

    fprintf (stderr, "\ntlmder starts ! \n");
    //  ...............................................................
    // choose the fastest GPU device
    //  ...............................................................
    unsigned int GPU_ID = 1; 
    // unsigned int GPU_ID =  cutGetMaxGflopsDeviceId() ;
    hipSetDevice(GPU_ID); 
    fprintf (stderr, " CUDA device chosen = %d \n", GPU_ID);

    // ....................................................... 
    //  get memory in the GPU to store the results 
    // ....................................................... 
    ResultType * results_GPU = 0;
    cutilSafeCall(hipMalloc( &results_GPU,  NUM_THREADS * sizeof(ResultType) ));

    // ....................................................... 
    //  get memory in the CPU to store the results 
    // ....................................................... 
    ResultType * results_CPU = 0;
    cutilSafeCall(hipHostMalloc( &results_CPU, NUM_THREADS * sizeof(ResultType) ));

    // ....................................................... 
    //  launch the kernel
    // ....................................................... 
    fprintf (stderr, " \nlaunching the kernel num. blocks = %d, threads per block = %d\n total threads = %d\n\n",
             NUM_BLOCKS, NUM_THREADS_PER_BLOCK, NUM_THREADS);

    mainKernel<<<NUM_BLOCKS,NUM_THREADS_PER_BLOCK>>> ( results_GPU );

    // ....................................................... 
    // wait for termination
    // ....................................................... 
    hipDeviceSynchronize(); 
    fprintf (stderr, " GPU processing done \n\n");

    // ....................................................... 
    // copy back to CPU the results
    // ....................................................... 
    cutilSafeCall(hipMemcpy( results_CPU, results_GPU,
                              NUM_THREADS * sizeof(ResultType),
                              hipMemcpyDeviceToHost
                              ));

    // ....................................................... 
    // check all the threads computed the same results
    // ....................................................... 
    bool ok = true;
    for (unsigned int i = 1; i<NUM_THREADS; i++) {
	if ( memcmp (&results_CPU[0], &results_CPU[i], sizeof(ResultType)) != 0) {
            // warning: may the padding bytes be different ?
            ok = false;
	}
    } // for

    if (ok) {
	fprintf (stderr, " !!! all threads computed the same results !!! \n\n");
    } else {
	fprintf (stderr, "ERROR in results of threads \n");
    }

    // ....................................................... 
    // show the results !
    // ....................................................... 

    printf("      final l2 norm of the residuals%15.7g\n\n", results_CPU[0].fnorm);
    printf("      number of function evaluations%10i\n\n", results_CPU[0].nfev);
    printf("      number of Jacobian evaluations%10i\n\n", results_CPU[0].njev);
    printf("      exit parameter                %10i\n\n", results_CPU[0].info);
    printf("      final approximate solution\n");

    for (int j=0; j<NUM_PARAMS; j++)  {
        printf("%15.7g", results_CPU[0].solution[j]);
    }
    printf("\n");

    printf("      covariance\n");
 
    for (unsigned int i=0; i<NUM_PARAMS; i++) {
        for (unsigned int j=0; j<NUM_PARAMS; j++) {
            printf("%15.7g", results_CPU[0].covar[i][j]);
	} // for
	printf ("\n");
    } // for
  
    printf("\n");
    printf(" rank(J) = %d\n", results_CPU[0].rankJ );

    cutilSafeCall(hipFree(results_GPU));
    cutilSafeCall(hipHostFree(results_CPU));
    hipDeviceReset();
    //cutilExit(argc, argv);
} // ()
