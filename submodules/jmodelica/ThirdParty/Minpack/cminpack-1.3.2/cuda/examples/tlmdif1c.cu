#include "hip/hip_runtime.h"
/* -*- mode: c++ -*- */
/* ------------------------------ */
/*  driver for lmdif1 example.     */
/* ------------------------------ */

#include <stdio.h>
#include <math.h>
#include <string.h>
#include <cminpack.h>

#include <lmdif1.cu>
#define real __cminpack_real__

#define cutilSafeCall(err)           __cudaSafeCall      (err, __FILE__, __LINE__)
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    if( hipSuccess != err) {
        fprintf(stderr, "cudaSafeCall() Runtime API error in file <%s>, line %i : %s.\n",
                file, line, hipGetErrorString( err) );
        exit(-1);
    }
}

//--------------------------------------------------------------------------
//--------------------------------------------------------------------------
const unsigned int NUM_OBSERVATIONS = 15; // m
const unsigned int NUM_PARAMS = 3; // 3 = n

//--------------------------------------------------------------------------
//--------------------------------------------------------------------------
// 
//  fixed arrangement of threads to be run 
// 
const unsigned int NUM_THREADS = 2048;
const unsigned int NUM_THREADS_PER_BLOCK = 128;
const unsigned int NUM_BLOCKS = NUM_THREADS / NUM_THREADS_PER_BLOCK;

//--------------------------------------------------------------------------
//--------------------------------------------------------------------------
//
// the struct for returning results from the GPU
//

typedef struct    
{
    real fnorm;
    int info;
    real solution[NUM_PARAMS];
} ResultType;


//--------------------------------------------------------------------------
// the cost function
//--------------------------------------------------------------------------
__cminpack_attr__ /* __device__ */
int fcn_mn(void *p, int m, int n, const real *x, real *fvec, int iflag)
{

    /*      subroutine fcn for lmdif example. */

    int i;
    real tmp1, tmp2, tmp3;
    real y[NUM_OBSERVATIONS]={1.4e-1, 1.8e-1, 2.2e-1, 2.5e-1, 2.9e-1, 3.2e-1, 3.5e-1,
                              3.9e-1, 3.7e-1, 5.8e-1, 7.3e-1, 9.6e-1, 1.34, 2.1, 4.39};

    for (i = 0; i < NUM_OBSERVATIONS; i++) {
        tmp1 = i+1;
        tmp2 = NUM_OBSERVATIONS - i;
        tmp3 = tmp1;
	  
        if (i >= 8) tmp3 = tmp2;
        fvec[i] = y[i] - (x[0] + tmp1/(x[1]*tmp2 + x[2]*tmp3));
    } // for
    return 0;
}

//--------------------------------------------------------------------------
// the kernel in the GPU
//--------------------------------------------------------------------------
__global__ void mainKernel(ResultType  pResults[])
{

    int info, lwa, iwa[NUM_PARAMS];
    real tol, fnorm, x[NUM_PARAMS], fvec[NUM_OBSERVATIONS], wa[75];

    // m = 15 = NUM_OBSERVATIONS;
    // n = 3 = NUM_PARAMS;

    /*   the following starting values provide a rough fit. */
    x[0] = 1.; 
    x[1] = 1.; 
    x[2] = 1.;

    lwa = 75;

    /* set tol to the square root of the machine precision.  unless high
       precision solutions are required, this is the recommended
       setting. */

    tol = sqrt(__cminpack_func__(dpmpar)(1));

    // -------------------------------
    // call lmdif, and enorm
    // -------------------------------
    info = __cminpack_func__(lmdif1)(__cminpack_param_fcn_mn__ 0, NUM_OBSERVATIONS, NUM_PARAMS, x, fvec, tol, iwa, wa, lwa);
  
    fnorm = __cminpack_func__(enorm)(NUM_OBSERVATIONS, fvec);

    // ----------------------------------
    // save the results in global memory
    // ----------------------------------
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;

    pResults[threadId].fnorm = fnorm;
    pResults[threadId].info = info;

    for (int j=0; j<NUM_PARAMS; j++) {
        pResults[threadId].solution[j] = x[j];
    }

} // ()

//--------------------------------------------------------------------------
//--------------------------------------------------------------------------
int main (int argc, char** argv)
{

    fprintf (stderr, "\ntlmdif1c starts ! \n");

    //  ...............................................................
    // choose the fastest GPU device
    //  ...............................................................
    unsigned int GPU_ID = 1;  // not actually :-)
    // unsigned int GPU_ID =  cutGetMaxGflopsDeviceId() ;
    hipSetDevice(GPU_ID); 
    fprintf (stderr, " CUDA device chosen = %d \n", GPU_ID);

    // ....................................................... 
    //  get memory in the GPU to store the results 
    // ....................................................... 
    ResultType * results_GPU = 0;
    cutilSafeCall( hipMalloc( &results_GPU,  NUM_THREADS * sizeof(ResultType)) );

    // ....................................................... 
    //  get memory in the CPU to store the results 
    // ....................................................... 
    ResultType * results_CPU = 0;
    cutilSafeCall( hipHostMalloc( &results_CPU, NUM_THREADS * sizeof(ResultType)) );

    // ....................................................... 
    //  launch the kernel
    // ....................................................... 
    fprintf (stderr, " \nlaunching the kernel num. blocks = %d, threads per block = %d\n total threads = %d\n\n",
             NUM_BLOCKS, NUM_THREADS_PER_BLOCK, NUM_THREADS);

    mainKernel<<<NUM_BLOCKS,NUM_THREADS_PER_BLOCK>>> ( results_GPU );

    // ....................................................... 
    // wait for termination
    // ....................................................... 
    hipDeviceSynchronize(); 
    fprintf (stderr, " GPU processing done \n\n");

    // ....................................................... 
    // copy back to CPU the results
    // ....................................................... 
    cutilSafeCall( hipMemcpy( results_CPU, results_GPU, 
                               NUM_THREADS * sizeof(ResultType),
                               hipMemcpyDeviceToHost
                               ) );

    // ....................................................... 
    // check all the threads computed the same results
    // ....................................................... 
    bool ok = true;
    for (unsigned int i = 1; i<NUM_THREADS; i++) {
	if ( memcmp (&results_CPU[0], &results_CPU[i], sizeof(ResultType)) != 0) {
            // warning: may the padding bytes be different ?
            ok = false;
	}
    } // for
		 

    if (ok) {
	fprintf (stderr, " !!! all threads computed the same results !!! \n\n");
    } else {
	fprintf (stderr, "ERROR in results of threads \n");
    }

    // ....................................................... 
    // show the results !
    // ....................................................... 

    printf("      final l2 norm of the residuals%15.7g\n\n", results_CPU[0].fnorm);
    printf("      exit parameter                %10i\n\n", results_CPU[0].info);
    printf("      final approximate solution\n");

    for (int j=0; j<NUM_PARAMS; j++)  {
        printf("%15.7g", results_CPU[0].solution[j]);
    }
    printf("\n");

    cutilSafeCall(hipFree(results_GPU));
    cutilSafeCall(hipHostFree(results_CPU));
    hipDeviceReset();
    //cutilExit(argc, argv);
} // ()

