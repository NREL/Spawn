#include "hip/hip_runtime.h"
/* -*- mode: c++ -*- */
/* ------------------------------ */
/*  driver for lmdif example.     */
/* ------------------------------ */

#include <stdio.h>
#include <math.h>
#include <string.h>
#include <cminpack.h>

#include <lmdif.cu>
#include <covar1.cu>
#define real __cminpack_real__

#define cutilSafeCall(err)           __cudaSafeCall      (err, __FILE__, __LINE__)
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    if( hipSuccess != err) {
        fprintf(stderr, "cudaSafeCall() Runtime API error in file <%s>, line %i : %s.\n",
                file, line, hipGetErrorString( err) );
        exit(-1);
    }
}

//--------------------------------------------------------------------------
//--------------------------------------------------------------------------
const unsigned int NUM_OBSERVATIONS = 15; // m
const unsigned int NUM_PARAMS = 3; // 3 = n

//--------------------------------------------------------------------------
//--------------------------------------------------------------------------
// 
//  fixed arrangement of threads to be run 
// 
const unsigned int NUM_THREADS = 2048;
const unsigned int NUM_THREADS_PER_BLOCK = 128;
const unsigned int NUM_BLOCKS = NUM_THREADS / NUM_THREADS_PER_BLOCK;

//--------------------------------------------------------------------------
//--------------------------------------------------------------------------
//
// the struct for returning results from the GPU
//
//   #define ALIGN 16
// #pragma pack(16)
// float rankJ;
// __align__(2*sizeof(float))
// __align__(ALIGN) 
// float covar[NUM_PARAMS][NUM_PARAMS];


typedef struct    
{
    real fnorm;
    int nfev;
    int info;
    int rankJ;
    real solution[NUM_PARAMS];
    real covar[NUM_PARAMS][NUM_PARAMS];
} ResultType;

//--------------------------------------------------------------------------
// the cost function
//--------------------------------------------------------------------------
__cminpack_attr__ /* __device__ */
int fcn_mn(void *p, int m, int n, const real *x, real *fvec, int iflag)
{

    /*      subroutine fcn for lmdif example. */

    int i;
    real tmp1, tmp2, tmp3;
    real y[NUM_OBSERVATIONS]={1.4e-1, 1.8e-1, 2.2e-1, 2.5e-1, 2.9e-1, 3.2e-1, 3.5e-1,
                              3.9e-1, 3.7e-1, 5.8e-1, 7.3e-1, 9.6e-1, 1.34, 2.1, 4.39};

    if (iflag == 0)
    {
        /*      insert print statements here when nprint is positive. */
        return 0;
    }
    for (i = 1; i <= NUM_OBSERVATIONS; i++)
    {
        tmp1 = i;
        tmp2 = (NUM_OBSERVATIONS+1) - i;
        tmp3 = tmp1;
        if (i > 8) tmp3 = tmp2;
        fvec[i-1] = y[i-1] - (x[1-1] + tmp1/(x[2-1]*tmp2 + x[3-1]*tmp3));
    }
    return 0;
}

//--------------------------------------------------------------------------
// a test kernel for cheking the return of results
//--------------------------------------------------------------------------
__global__ void test_mainKernel(ResultType * pResults)
{

    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;

    pResults[threadId].fnorm = 13.13;
    pResults[threadId].nfev = 144;
    pResults[threadId].info = -1234;

    for (int j=0; j<NUM_PARAMS; j++) {
	pResults[threadId].solution[j] = 200+sqrt((real) (j*j));
    }

    for (unsigned int i=0; i<NUM_PARAMS; i++) {
        for (unsigned int j=0; j<NUM_PARAMS; j++) {
            pResults[threadId].covar[i][j] = 100+i+j;
	} // for
    } // for

    pResults[threadId].rankJ =   NUM_PARAMS; 

} // ()

//--------------------------------------------------------------------------
// the kernel in the GPU
//--------------------------------------------------------------------------
__global__ void mainKernel(ResultType  pResults[])
{
    int  maxfev, mode, nprint, info, nfev, ldfjac;
    int ipvt[NUM_PARAMS];
    real ftol, xtol, gtol, epsfcn, factor, fnorm;
    real x[NUM_PARAMS], fvec[NUM_OBSERVATIONS], 
            diag[NUM_PARAMS], fjac[NUM_OBSERVATIONS*NUM_PARAMS], qtf[NUM_PARAMS], 
            wa1[NUM_PARAMS], wa2[NUM_PARAMS], wa3[NUM_PARAMS], wa4[NUM_OBSERVATIONS];
    int k;

    // m = NUM_OBSERVATIONS;
    // n = NUM_PARAMS;

    /*      the following starting values provide a rough fit. */

    x[1-1] = 1.; 
    x[2-1] = 1.; 
    x[3-1] = 1.;

    ldfjac = NUM_OBSERVATIONS;

    /*      set ftol and xtol to the square root of the machine */
    /*      and gtol to zero. unless high solutions are */
    /*      required, these are the recommended settings. */

    ftol = sqrt(__cminpack_func__(dpmpar)(1));
    xtol = sqrt(__cminpack_func__(dpmpar)(1));
    gtol = 0.;

    maxfev = 800;
    epsfcn = 0.;
    mode = 1;
    factor = 1.e2;
    nprint = 0;

    /* NOTE: lmdif for pointer to cost function
       Error: Function pointers and function template parameters are not supported in sm_1x.
       info = lmdif(COST_FUNCTION, 0, m, n, x, fvec, ftol, xtol, gtol, maxfev, epsfcn, 
       diag, mode, factor, nprint, &nfev, fjac, ldfjac, 
       ipvt, qtf, wa1, wa2, wa3, wa4);
    */

    // -------------------------------
    // call lmdif, enorm, and covar1
    // -------------------------------
    info = __cminpack_func__(lmdif)(__cminpack_param_fcn_mn__ 0, NUM_OBSERVATIONS, NUM_PARAMS, 
                 x, fvec, ftol, xtol, gtol, maxfev, epsfcn, 
                 diag, mode, factor, nprint, &nfev, fjac, ldfjac, 
                 ipvt, qtf, wa1, wa2, wa3, wa4);

    fnorm = __cminpack_func__(enorm)(NUM_OBSERVATIONS, fvec);

    // NOTE: REMOVED THE TEST OF ORIGINAL MINPACK covar routine

    /* test covar1, which also estimates the rank of the Jacobian */
    ftol = __cminpack_func__(dpmpar)(1);
    k = __cminpack_func__(covar1)(NUM_OBSERVATIONS, NUM_PARAMS, 
               fnorm*fnorm, fjac, ldfjac, ipvt, ftol, wa1);

    // ----------------------------------
    // save the results in global memory
    // ----------------------------------
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;

    pResults[threadId].fnorm = fnorm;
    pResults[threadId].nfev = nfev;
    pResults[threadId].info = info;

    for (int j=1; j<=NUM_PARAMS; j++) {
        pResults[threadId].solution[j-1] = x[j-1];
    }

    for (int i=1; i<=NUM_PARAMS; i++) {
        for (int j=1; j<=NUM_PARAMS; j++) {
            pResults[threadId].covar[i-1][j-1] = fjac[(i-1)*ldfjac+j-1];
	} // for
    } // for

    /*
      for (unsigned int i=0; i<NUM_PARAMS; i++) {
      for (unsigned int j=0; j<NUM_PARAMS; j++) {
      pResults[threadId].covar[i][j] = 100+i+j;
      } // for
      } // for
    */

    pResults[threadId].rankJ =  (k != 0 ? k : NUM_PARAMS); 

} // ()

//--------------------------------------------------------------------------
//--------------------------------------------------------------------------
int main (int argc, char** argv)
{

    fprintf (stderr, "\ntlmdif starts ! \n");
    //  ...............................................................
    // choose the fastest GPU device
    //  ...............................................................
    unsigned int GPU_ID = 1; 
    //unsigned int GPU_ID =  cutGetMaxGflopsDeviceId() ;
    hipSetDevice(GPU_ID); 
    fprintf (stderr, " CUDA device chosen = %d \n", GPU_ID);

    // ....................................................... 
    //  get memory in the GPU to store the results 
    // ....................................................... 
    ResultType * results_GPU = 0;
    cutilSafeCall( hipMalloc( &results_GPU,  NUM_THREADS * sizeof(ResultType) ) );

    // ....................................................... 
    //  get memory in the CPU to store the results 
    // ....................................................... 
    ResultType * results_CPU = 0;
    cutilSafeCall( hipHostMalloc( &results_CPU, NUM_THREADS * sizeof(ResultType) ) );

    // ....................................................... 
    //  launch the kernel
    // ....................................................... 
    fprintf (stderr, " \nlaunching the kernel num. blocks = %d, threads per block = %d\n total threads = %d\n\n",
             NUM_BLOCKS, NUM_THREADS_PER_BLOCK, NUM_THREADS);

    mainKernel<<<NUM_BLOCKS,NUM_THREADS_PER_BLOCK>>> ( results_GPU );

    // ....................................................... 
    // wait for termination
    // ....................................................... 
    hipDeviceSynchronize(); 
    fprintf (stderr, " GPU processing done \n\n");

    // ....................................................... 
    // copy back to CPU the results
    // ....................................................... 
    cutilSafeCall( hipMemcpy( results_CPU, results_GPU, 
                               NUM_THREADS * sizeof(ResultType),
                               hipMemcpyDeviceToHost
                               ) );

    // ....................................................... 
    // check all the threads computed the same results
    // ....................................................... 
    bool ok = true;
    for (unsigned int i = 1; i<NUM_THREADS; i++) {
	if ( memcmp (&results_CPU[0], &results_CPU[i], sizeof(ResultType)) != 0) {
            // warning: may the padding bytes be different ?
            ok = false;
	}
	/*
          if ( results_CPU[0].fnorm != results_CPU[i].fnorm
          || results_CPU[0].nfev != results_CPU[i].nfev
          || results_CPU[0].info != results_CPU[i].info
          || results_CPU[0].rankJ != results_CPU[i].rankJ
          )
	  {
          ok = false;
	  }
	*/
    } // for
		 

    if (ok) {
	fprintf (stderr, " !!! all threads computed the same results !!! \n\n");
    } else {
	fprintf (stderr, "ERROR in results of threads \n");
    }

    // ....................................................... 
    // show the results !
    // ....................................................... 

    printf("      final l2 norm of the residuals%15.7g\n\n", results_CPU[0].fnorm);
    printf("      number of function evaluations%10i\n\n", results_CPU[0].nfev);
    printf("      exit parameter                %10i\n\n", results_CPU[0].info);
    printf("      final approximate solution\n");

    for (int j=0; j<NUM_PARAMS; j++)  {
        printf("%15.7g", results_CPU[0].solution[j]);
    }
    printf("\n");

    printf("      covariance\n");

 
    for (unsigned int i=0; i<NUM_PARAMS; i++) {
        for (unsigned int j=0; j<NUM_PARAMS; j++) {
            printf("%15.7g", results_CPU[0].covar[i][j]);
	} // for
	printf ("\n");
    } // for
  
    printf("\n");
    printf(" rank(J) = %d\n", results_CPU[0].rankJ );
    cutilSafeCall(hipFree(results_GPU));
    cutilSafeCall(hipHostFree(results_CPU));
    hipDeviceReset();
    //cutilExit(argc, argv);

} // ()
